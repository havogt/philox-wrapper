#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "philox-wrapper/PhiloxWrapper.h"

__global__ void test_kernel(int rngSeed, int rngCounter) {
    philox_wrapper::PhiloxWrapper<double> rng(
        blockIdx.x * blockDim.x + threadIdx.x, rngSeed, rngCounter);
    printf("%f\n", rng.rand2());
}

TEST(Stupid, Example) {
    int seed = 123;
    for (size_t i = 0; i < 5; ++i) {
        test_kernel<<<1, 1>>>(
            seed, philox_wrapper::PhiloxWrapper<double>::getNextCounter());
    }
    hipDeviceSynchronize();
}
